#include "CuContext.cuh"

#ifdef CRIA_PACO_CUDA 

namespace cria_ai { namespace paco { namespace cu {
	
	crresult CRCuContext::init()
	{
		return CRRES_OK;
	}

	void* CRCuMalloc(size_t size)
	{
		void* mem = nullptr;
		hipError_t res = hipMallocManaged(&mem, size);
		
		CRIA_AUTO_ASSERT(res == hipSuccess, "Target size: %llu, Cuda Error code: %i", size, res);
		if (res != hipSuccess)
			return nullptr;

		return mem;
	}
	void CRCuFree(void* mem)
	{
		if (mem)
			hipFree(mem);
	}
}}}

#endif //CRIA_PACO_CUDA 
