#include "CuContext.cuh"

#ifdef CRIA_PACO_CUDA 

namespace cria_ai { namespace paco { namespace cu {
	
	crresult CRCuContext::init()
	{
		return CRRES_OK;
	}

	void* CRCuMalloc(size_t size)
	{
		void* mem = nullptr;
		hipMallocManaged(&mem, size);
		return mem;
	}
	void CRCuFree(void* mem)
	{
		if (mem)
			hipFree(mem);
	}
}}}

#endif //CRIA_PACO_CUDA 
