#include "hip/hip_runtime.h"
#include "../ActivationFunctions.h"

#ifdef CRIA_PACO_CUDA 

#include "CuContext.cuh"

#define CR_CUDA_AF_BLOCK_COUNT         1
#define CR_CUDA_AF_THREAD_COUNT        256

namespace cria_ai { namespace paco {
	
	/**
	* \brief A activation function
	*
	* Equation:     1 / (1 + e^-x) = r
	* Output Range: (0 < x < 1)
	*
	* \param input  A matrix containing values for processing.
	* \param output A matrix that holds the output values.
	*/
	__global__ void CRCuSigmoid(CRNWMat const* input, CRNWMat* output);
	void CRSigmoid(CRNWMat const* input, CRNWMat* output)
	{
		CRIA_SIGMOID_VALIDATION_CHECK(input, output);

		CRCuSigmoid<<<CR_CUDA_AF_BLOCK_COUNT, CR_CUDA_AF_THREAD_COUNT >>>(input, output);
		hipDeviceSynchronize();
	}
	__global__ void CRCuSigmoid(CRNWMat const* input, CRNWMat* output)
	{
		int startIndex = blockIdx.x * blockDim.x + threadIdx.x;
		int stride = blockDim.x * gridDim.x;

		for (uint index = startIndex; index < input->Cols * input->Rows; index += stride)
		{
			output->Data[index] = 1.0f / (1.0f + __expf(-input->Data[index]));
		}
	}

	/**
	* \brief A inverse activation function
	*
	* Equation:    -ln((1/r) - 1) = x
	* Input Range: (0 < r < 1)
	*
	* \param input  A matrix containing values for processing.
	* \param output A matrix that holds the output values.
	*/
	__global__ void CRCuSigmoidInv(CRNWMat const* input, CRNWMat* output);
	void CRSigmoidInv(CRNWMat const* input, CRNWMat* output)
	{
		CRIA_SIGMOID_VALIDATION_CHECK(input, output);

		CRCuSigmoidInv<<<CR_CUDA_AF_BLOCK_COUNT, CR_CUDA_AF_THREAD_COUNT>>>(input, output);
		hipDeviceSynchronize();
	}
	__global__ void CRCuSigmoidInv(CRNWMat const* input, CRNWMat* output)
	{
		int startIndex = blockIdx.x * blockDim.x + threadIdx.x;
		int stride = blockDim.x * gridDim.x;

		for (uint index = startIndex; index < input->Cols * input->Rows; index += stride) {

			if (input->Data[index] > 0 || input->Data[index] < 1)
				output->Data[index] = -__logf((1 / input->Data[index]) - 1);
			else 
				output->Data[index] = 0;

		}
	}
}}

#endif //CRIA_PACO_CUDA 
